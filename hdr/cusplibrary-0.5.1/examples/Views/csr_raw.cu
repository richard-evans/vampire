#include <cusp/csr_matrix.h>
#include <cusp/print.h>
#include <cusp/multiply.h>

// This example shows how to wrap "raw" host and device memory
// with a csr_matrix_view.  This situation arises when interfacing
// Cusp with data that is managed externally.  Once raw data has
// been appropriately wrapped the csr_matrix_view can be used
// in Cusp algorithms like cusp::copy() and cusp::multiply()
// just like a csr_matrix container.
//
//  Example Matrix:
//   [10  0 20]
//   [ 0  0  0]
//   [ 0  0 30]
//   [40 50 60]

int main(void)
{
  // CSR format in raw host memory
  int   host_Ap[5] = {0,2,2,3,6};            // CSR row pointer
  int   host_Aj[6] = {0,2,2,0,1,2};          // CSR column indices
  float host_Ax[6] = {10,20,30,40,50,60};    // CSR values

  // x and y arrays in host memory
  float host_x[3] = {1,1,1};
  float host_y[4] = {0,0,0,0};

  // allocate device memory for CSR format
  int   * device_Ap;  hipMalloc(&device_Ap, 5 * sizeof(int));
  int   * device_Aj;  hipMalloc(&device_Aj, 6 * sizeof(int));
  float * device_Ax;  hipMalloc(&device_Ax, 6 * sizeof(float));
  
  // allocate device memory for x and y arrays
  float * device_x;   hipMalloc(&device_x, 3 * sizeof(float));
  float * device_y;   hipMalloc(&device_y, 4 * sizeof(float));

  // copy raw data from host to device
  hipMemcpy(device_Ap, host_Ap, 5 * sizeof(int),   hipMemcpyHostToDevice);
  hipMemcpy(device_Aj, host_Aj, 6 * sizeof(int),   hipMemcpyHostToDevice);
  hipMemcpy(device_Ax, host_Ax, 6 * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_x,  host_x,  3 * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_y,  host_y,  4 * sizeof(float), hipMemcpyHostToDevice);

  // wrap the host memory with a csr_matrix_view
  {
    // use array1d_view to wrap the individual arrays
    typedef typename cusp::array1d_view<int   *> HostIndexArrayView;
    typedef typename cusp::array1d_view<float *> HostValueArrayView;

    HostIndexArrayView row_offsets   (host_Ap, host_Ap + 5);
    HostIndexArrayView column_indices(host_Aj, host_Aj + 6);
    HostValueArrayView values        (host_Ax, host_Ax + 6);
    
    HostValueArrayView x (host_x, host_x + 3);
    HostValueArrayView y (host_y, host_y + 4);

    // combine the three array1d_views into a csr_matrix_view
    typedef cusp::csr_matrix_view<HostIndexArrayView,
                                  HostIndexArrayView,
                                  HostValueArrayView> HostView;

    HostView A(4, 3, 6, row_offsets, column_indices, values);

    // print view
    std::cout << "\nhost csr_matrix_view" << std::endl;
    cusp::print(A);

    // compute y = A* x 
    cusp::multiply(A, x, y);
    
    // print x
    std::cout << "\nx array" << std::endl;
    cusp::print(x);

    // print y
    std::cout << "\n y array, y = A * x" << std::endl;
    cusp::print(y);
  }

  // wrap the device memory with a csr_matrix_view
  {
    // *NOTE* raw pointers must be wrapped with thrust::device_ptr!
    thrust::device_ptr<int>   wrapped_device_Ap(device_Ap);
    thrust::device_ptr<int>   wrapped_device_Aj(device_Aj);
    thrust::device_ptr<float> wrapped_device_Ax(device_Ax);
    thrust::device_ptr<float> wrapped_device_x(device_x);
    thrust::device_ptr<float> wrapped_device_y(device_y);

    // use array1d_view to wrap the individual arrays
    typedef typename cusp::array1d_view< thrust::device_ptr<int>   > DeviceIndexArrayView;
    typedef typename cusp::array1d_view< thrust::device_ptr<float> > DeviceValueArrayView;

    DeviceIndexArrayView row_offsets   (wrapped_device_Ap, wrapped_device_Ap + 5);
    DeviceIndexArrayView column_indices(wrapped_device_Aj, wrapped_device_Aj + 6);
    DeviceValueArrayView values        (wrapped_device_Ax, wrapped_device_Ax + 6);
    DeviceValueArrayView x (wrapped_device_x, wrapped_device_x + 3);
    DeviceValueArrayView y (wrapped_device_y, wrapped_device_y + 4);

    // combine the three array1d_views into a csr_matrix_view
    typedef cusp::csr_matrix_view<DeviceIndexArrayView,
                                  DeviceIndexArrayView,
                                  DeviceValueArrayView> DeviceView;

    DeviceView A(4, 3, 6, row_offsets, column_indices, values);

    // print view
    std::cout << "\ndevice csr_matrix_view" << std::endl;
    cusp::print(A);

    // compute y = A* x 
    cusp::multiply(A, x, y);
    
    // print x
    std::cout << "\nx array" << std::endl;
    cusp::print(x);

    // print y
    std::cout << "\n y array, y = A * x" << std::endl;
    cusp::print(y);
  }
  
  // free device arrays
  hipFree(device_Ap);
  hipFree(device_Aj);
  hipFree(device_Ax);
  hipFree(device_x);
  hipFree(device_y);

  return 0;
}

