#include "hip/hip_runtime.h"
#include <cusp/linear_operator.h>
#include <cusp/krylov/cg.h>

// This example shows how to use cusp::linear_operator to solve
// a linear system with a user-defined linear operator A.  The
// linear_operator is a way to interface custom sparse matrix
// formats or so-called "matrix-free" methods with the iterative
// solvers in Cusp.  In this example, we illustrate a matrix-free
// implementation of a simple 5-point finite-difference stencil,
//
//                [  0 -1  0 ]
//                [ -1  4 -1 ]
//                [  0 -1  0 ]
//
// using a CUDA kernel.  We combine the linear_operator with the
// Conjugate Gradient method to solve a 2D Poisson problem.


__global__
void stencil_kernel(int N, const float * x, float * y)
{
    // compute y = A*x, where A is the 5-point stencil
    // note: pre-caching a window of x into __shared__ memory
    // would make this a lot faster.

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if (i < N && j < N)
    {
        // linear index into 2D grid
        int index = N * i + j;

        float result = 4.0f * x[index];         // center point

        if (i > 0    ) result -= x[index - N];  // lower neighbor
        if (i < N - 1) result -= x[index + N];  // upper neighbor
        if (j > 0    ) result -= x[index - 1];  // left neighbor
        if (j < N - 1) result -= x[index + 1];  // right neighbor

        // write result
        y[N * i + j] = result;
    }
}

class stencil : public cusp::linear_operator<float,cusp::device_memory>
{
public:
    typedef cusp::linear_operator<float,cusp::device_memory> super;

    int N;

    // constructor
    stencil(int N)
        : super(N*N,N*N), N(N) {}

    // linear operator y = A*x
    template <typename VectorType1,
             typename VectorType2>
    void operator()(const VectorType1& x, VectorType2& y) const
    {
        // obtain a raw pointer to device memory
        const float * x_ptr = thrust::raw_pointer_cast(&x[0]);
        float * y_ptr = thrust::raw_pointer_cast(&y[0]);

        dim3 dimBlock(16,16);
        dim3 dimGrid((N + 15) / 16, (N + 15) / 16);

        stencil_kernel<<<dimGrid,dimBlock>>>(N, x_ptr, y_ptr);
    }
};


int main(void)
{
    // number of grid points in each dimension
    const int N = 10;

    // create a matrix-free linear operator
    stencil A(N);

    // allocate storage for solution (x) and right hand side (b)
    cusp::array1d<float, cusp::device_memory> x(A.num_rows, 0);
    cusp::array1d<float, cusp::device_memory> b(A.num_rows, 1);

    // set stopping criteria:
    //  iteration_limit    = 100
    //  relative_tolerance = 1e-6
    cusp::monitor<float> monitor(b, 100, 1e-5, 0, false);

    // solve the linear system A * x = b with the Conjugate Gradient method
    cusp::krylov::cg(A, x, b, monitor);

    return 0;
}

