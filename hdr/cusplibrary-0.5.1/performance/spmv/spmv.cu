#include <cusp/csr_matrix.h>
#include <cusp/io/matrix_market.h>
#include <cusp/gallery/poisson.h>

#include <iostream>
#include <string>
#include <map>
#include <cmath>
#include <limits>

#include <cusp/multiply.h>

#include "bytes_per_spmv.h"
#include "utility.h"
#include "benchmark.h"

typedef std::map<std::string, std::string> ArgumentMap;
ArgumentMap args;

std::string process_args(int argc, char ** argv)
{
    std::string filename;

    for(int i = 1; i < argc; i++)
    {
        std::string arg(argv[i]);

        if (arg.substr(0,2) == "--")
        {
            std::string::size_type n = arg.find('=',2);

            if (n == std::string::npos)
                args[arg.substr(2)] = std::string();              // (key)
            else
                args[arg.substr(2, n - 2)] = arg.substr(n + 1);   // (key,value)
        }
        else
        {
            filename = arg;
        }
    }

    return filename;
}

void usage(int argc, char** argv)
{
    std::cout << "Usage:\n";
    std::cout << "\t" << argv[0] << "\n";
    std::cout << "\t" << argv[0] << " my_matrix.mtx\n";
    std::cout << "\t" << argv[0] << " my_matrix.mtx --device=1\n";
    std::cout << "\t" << argv[0] << " my_matrix.mtx --value_type=double\n\n";
    std::cout << "Note: my_matrix.mtx must be real-valued sparse matrix in the MatrixMarket file format.\n";
    std::cout << "      If no matrix file is provided then a simple example is created.\n";
}


template <typename IndexType, typename ValueType>
void test_all_formats(std::string& filename)
{
    int device_id  = args.count("device") ? atoi(args["device"].c_str()) :  0;
    set_device(device_id);
    list_devices();

    std::cout << "Running on Device " << device_id << "\n\n";

    // load a matrix stored in MatrixMarket format
    cusp::csr_matrix<IndexType, ValueType, cusp::host_memory> host_matrix;

    if (filename == "")
    {
        std::cout << "Generated matrix (poisson5pt) ";
        cusp::gallery::poisson5pt(host_matrix, 512, 512);
    }
    else
    {
        cusp::io::read_matrix_market_file(host_matrix, filename);
        std::cout << "Read matrix (" << filename << ") ";
    }

    std::cout << "with shape ("  << host_matrix.num_rows << "," << host_matrix.num_cols << ") and "
              << host_matrix.num_entries << " entries" << "\n\n";

    FILE * fid = fopen(BENCHMARK_OUTPUT_FILE_NAME, "a");
    fprintf(fid, "file=%s rows=%d cols=%d nonzeros=%d\n", filename.c_str(),
            (int) host_matrix.num_rows, (int) host_matrix.num_cols, (int) host_matrix.num_entries);
    fclose(fid);

    test_coo(host_matrix);
    test_csr(host_matrix);
    test_dia(host_matrix);
    test_ell(host_matrix);
    test_hyb(host_matrix);
}

int main(int argc, char** argv)
{
    std::string filename = process_args(argc, argv);

    if (args.count("help"))
    {
        usage(argc, argv);
        return 0;
    }

    // select ValueType
    std::string value_type = args.count("value_type") ? args["value_type"] : "float";
    std::cout << "\nComputing SpMV with \'" << value_type << "\' values.\n\n";

    if (value_type == "float")
    {
        test_all_formats<int,float>(filename);
    }
    else if (value_type == "double")
    {
        int current_device = -1;
        hipDeviceProp_t properties;
        hipGetDevice(&current_device);
        hipGetDeviceProperties(&properties, current_device);
        if (properties.major == 1 && properties.minor < 3)
            std::cerr << "ERROR: Support for \'double\' requires Compute Capability 1.3 or greater\n\n";
        else
            test_all_formats<int,double>(filename);
    }
    else
    {
        std::cerr << "ERROR: Unsupported type \'" << value_type << "\'\n\n";
    }

    return 0;
}

