#include "hip/hip_runtime.h"
#define CUSP_USE_TEXTURE_MEMORY

#include <cusp/dia_matrix.h>
#include <cusp/csr_matrix.h>
#include <cusp/io/matrix_market.h>

#include <thrust/sequence.h>
#include <thrust/fill.h>

#include <iostream>

#include <cusp/detail/device/spmv/csr_vector.h>
#include "../timer.h"

template <bool UseCache, unsigned int THREADS_PER_VECTOR, typename IndexType, typename ValueType>
void perform_spmv(const cusp::csr_matrix<IndexType,ValueType,cusp::device_memory>& csr, 
                  const ValueType * x, 
                        ValueType * y)
{
    const unsigned int VECTORS_PER_BLOCK  = 128 / THREADS_PER_VECTOR;
    const unsigned int THREADS_PER_BLOCK  = VECTORS_PER_BLOCK * THREADS_PER_VECTOR;

    //const unsigned int MAX_BLOCKS = MAX_THREADS / THREADS_PER_BLOCK;
    const unsigned int MAX_BLOCKS = 16 * 1024;
    const unsigned int NUM_BLOCKS = std::min(MAX_BLOCKS, static_cast<unsigned int>((csr.num_rows + (VECTORS_PER_BLOCK - 1)) / VECTORS_PER_BLOCK));
    
    if (UseCache)
        bind_x(x);

    cusp::detail::device::spmv_csr_vector_kernel<IndexType, ValueType, VECTORS_PER_BLOCK, THREADS_PER_VECTOR, UseCache> <<<NUM_BLOCKS, THREADS_PER_BLOCK>>> 
        (csr.num_rows,
         thrust::raw_pointer_cast(&csr.row_offsets[0]),
         thrust::raw_pointer_cast(&csr.column_indices[0]),
         thrust::raw_pointer_cast(&csr.values[0]),
         x, y);

    if (UseCache)
        unbind_x(x);
}
 
template <unsigned int ThreadsPerVector, typename IndexType, typename ValueType>
float benchmark_matrix(const cusp::csr_matrix<IndexType,ValueType,cusp::device_memory>& csr)
{
    const size_t num_iterations = 100;

    cusp::array1d<ValueType, cusp::device_memory> x(csr.num_cols);
    cusp::array1d<ValueType, cusp::device_memory> y(csr.num_rows);

    // warmup
    perform_spmv<true, ThreadsPerVector>(csr, thrust::raw_pointer_cast(&x[0]), thrust::raw_pointer_cast(&y[0]));

    // time several SpMV iterations
    timer t;
    for(size_t i = 0; i < num_iterations; i++)
        perform_spmv<true, ThreadsPerVector>(csr, thrust::raw_pointer_cast(&x[0]), thrust::raw_pointer_cast(&y[0]));
    hipDeviceSynchronize();

    float sec_per_iteration = t.seconds_elapsed() / num_iterations;
    float gflops = 2.0 * (csr.num_entries/sec_per_iteration) / 1e9;

    return gflops;
}


template <typename IndexType, typename ValueType>
void make_synthetic_example(const IndexType N, const IndexType D, 
                            cusp::csr_matrix<IndexType, ValueType, cusp::device_memory>& csr)
{
//    // create DIA matrix with D diagonal bands
//    const IndexType NNZ = N * D - (D * (D - 1)) / 2;
//    cusp::dia_matrix<IndexType, ValueType, cusp::host_memory> dia(N, N, NNZ, D, N);
//    thrust::sequence(dia.diagonal_offsets.begin(), dia.diagonal_offsets.end());
//    thrust::fill(dia.values.values.begin(), dia.values.values.end(), 1);

    // create ELL matrix with D diagonals
    cusp::ell_matrix<IndexType, ValueType, cusp::host_memory> ell(N, D, N * D, D);
    for(IndexType i = 0; i < N; i++)
    {
        for(IndexType j = 0; j < D; j++)
        {
            ell.column_indices(i,j) = j;
            ell.values(i,j) = 1.0;
        }
    }

    // convert to CSR
    csr = ell;
}


int main(int argc, char** argv)
{
    typedef int   IndexType;
    typedef float ValueType;
        

    if (argc == 1)
    {
        // matrix varies along rows, # of threads per vector varies along column
        printf("matrix      , nnz per row,       2,       4,       8,      16,      32,\n");

        const IndexType N = 320 * 1000;
        const IndexType max_diagonals = 64;

        for(IndexType D = 1; D <= max_diagonals; D++)
        {
            cusp::csr_matrix<IndexType, ValueType, cusp::device_memory> csr;
            make_synthetic_example(N, D, csr);
            printf("dense_%02d    ,    %8.2f,", (int) D, (float) csr.num_entries / (float) csr.num_rows);
            printf("  %5.4f,", benchmark_matrix< 2, IndexType, ValueType>(csr));
            printf("  %5.4f,", benchmark_matrix< 4, IndexType, ValueType>(csr));
            printf("  %5.4f,", benchmark_matrix< 8, IndexType, ValueType>(csr));
            printf("  %5.4f,", benchmark_matrix<16, IndexType, ValueType>(csr));
            printf("  %5.4f,", benchmark_matrix<32, IndexType, ValueType>(csr));
            printf("\n");
        }
    }
    else
    {
        // matrix varies along rows, # of threads per vector varies along column
        printf("matrix              , nnz per row,       2,       4,       8,      16,      32,\n");

        for(int i = 1; i < argc; i++)
        {
            cusp::csr_matrix<IndexType, ValueType, cusp::device_memory> csr;
            cusp::io::read_matrix_market_file(csr, std::string(argv[i]));
            printf("%20s,    %8.2f,", argv[i], (float) csr.num_entries / (float) csr.num_rows);
            printf("  %5.4f,", benchmark_matrix< 2, IndexType, ValueType>(csr));
            printf("  %5.4f,", benchmark_matrix< 4, IndexType, ValueType>(csr));
            printf("  %5.4f,", benchmark_matrix< 8, IndexType, ValueType>(csr));
            printf("  %5.4f,", benchmark_matrix<16, IndexType, ValueType>(csr));
            printf("  %5.4f,", benchmark_matrix<32, IndexType, ValueType>(csr));
            printf("\n");
        }
    }

    return 0;
}

