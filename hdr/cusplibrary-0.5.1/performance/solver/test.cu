#define CUSP_USE_TEXTURE_MEMORY

#include <cusp/csr_matrix.h>
#include <cusp/hyb_matrix.h>
#include <cusp/gallery/poisson.h>
#include <cusp/io/matrix_market.h>
#include <cusp/krylov/cg.h>

#include <iostream>

#include "../timer.h"

template <typename Matrix>
void benchmark_matrix(const Matrix& A)
{
    typedef typename Matrix::memory_space MemorySpace;
    typedef typename Matrix::value_type   ValueType;

    const size_t N = A.num_rows;

    cusp::array1d<ValueType, MemorySpace> x(N,0);
    cusp::array1d<ValueType, MemorySpace> b(N,1);

    cusp::monitor<ValueType> monitor(b, 2000, 1e-5);

    // time solver
    timer t;

    cusp::krylov::cg(A, x, b, monitor);

    float time = t.seconds_elapsed();

    hipDeviceSynchronize();

    if (monitor.converged())
        std::cout << "  Successfully converged";
    else
        std::cout << "  Failed to converge";
    std::cout << " after " << monitor.iteration_count() << " iterations." << std::endl;

    std::cout << "  Solver time " << time << " seconds (" << (1e3 * time / monitor.iteration_count()) << "ms per iteration)" << std::endl;
}


int main(int argc, char** argv)
{
    typedef int    IndexType;
    typedef double ValueType;

    typedef cusp::hyb_matrix<IndexType,ValueType,cusp::host_memory>   HostMatrix;
    typedef cusp::hyb_matrix<IndexType,ValueType,cusp::device_memory> DeviceMatrix;

    HostMatrix A;

    if (argc == 1)
    {
        std::cout << "Using default matrix (5-pt Laplacian stencil)" << std::endl;
        cusp::gallery::poisson5pt(A, 1000, 1000);
    }
    else
    {
        std::cout << "Reading matrix from file: " << argv[1] << std::endl;
        cusp::io::read_matrix_market_file(A, std::string(argv[1]));
    }

    std::cout << "Running solver on host..." << std::endl;
    benchmark_matrix(A);

    std::cout << "Running solver on device..." << std::endl;
    benchmark_matrix(DeviceMatrix(A));

    return 0;
}

