#include <unittest/unittest.h>

#include <cusp/transpose.h>

#include <cusp/array2d.h>
#include <cusp/coo_matrix.h>
#include <cusp/csr_matrix.h>
#include <cusp/dia_matrix.h>
#include <cusp/ell_matrix.h>
#include <cusp/hyb_matrix.h>

template <typename MatrixType>
void initialize_matrix(MatrixType& matrix)
{
    cusp::array2d<float, cusp::host_memory> D(4,3);

    D(0,0) = 10.25;
    D(0,1) = 11.00;
    D(0,2) =  0.00;
    D(1,0) =  0.00;
    D(1,1) =  0.00;
    D(1,2) = 12.50;
    D(2,0) = 13.75;
    D(2,1) =  0.00;
    D(2,2) = 14.00;
    D(3,0) =  0.00;
    D(3,1) = 16.50;
    D(3,2) =  0.00;

    matrix = D;
}

template <typename MatrixType>
void verify_result(const MatrixType& matrix)
{
    typedef typename MatrixType::value_type ValueType;

    ASSERT_EQUAL(matrix.num_rows,    3);
    ASSERT_EQUAL(matrix.num_cols,    4);

    cusp::array2d<ValueType, cusp::host_memory> dense(matrix);

    ASSERT_EQUAL(dense(0,0), 10.25);
    ASSERT_EQUAL(dense(0,1),  0.00);
    ASSERT_EQUAL(dense(0,2), 13.75);
    ASSERT_EQUAL(dense(0,3),  0.00);
    ASSERT_EQUAL(dense(1,0), 11.00);
    ASSERT_EQUAL(dense(1,1),  0.00);
    ASSERT_EQUAL(dense(1,2),  0.00);
    ASSERT_EQUAL(dense(1,3), 16.50);
    ASSERT_EQUAL(dense(2,0),  0.00);
    ASSERT_EQUAL(dense(2,1), 12.50);
    ASSERT_EQUAL(dense(2,2), 14.00);
    ASSERT_EQUAL(dense(2,3),  0.00);
}


template <typename Matrix1, typename Matrix2>
void TestTranspose(void)
{
    typedef typename Matrix1::view View1;
    typedef typename Matrix2::view View2;

    Matrix1 A;

    initialize_matrix(A);

    {
        Matrix2 At;
        cusp::transpose(A, At);
        verify_result(At);
    }
    {
        View1 V(A);
        Matrix2 At;
        cusp::transpose(V, At);
        verify_result(At);
    }

    Matrix2 At;
    cusp::transpose(A, At);

    {
        View2 Vt(At);
        cusp::transpose(A, Vt);
        verify_result(Vt);
    }
    {
        View1 V(A);
        View2 Vt(At);
        cusp::transpose(V, Vt);
        verify_result(Vt);
    }
}



///////////////////////
// Instantiate Tests //
///////////////////////
template <class Space>
void TestTransposeArray2dVariablePitch(void)
{
    typedef typename cusp::array2d<float, Space, cusp::row_major>    RowMajor;
    typedef typename cusp::array2d<float, Space, cusp::column_major> ColumnMajor;

    TestTranspose<RowMajor,    RowMajor>();
    TestTranspose<ColumnMajor, ColumnMajor>();
    TestTranspose<RowMajor,    ColumnMajor>();
    TestTranspose<ColumnMajor, RowMajor>();

    // test with non-trivial pitch
    {
        RowMajor A(4,3);
        A.resize(4,3,5);
        A(0,0) = 10.25;
        A(0,1) = 11.00;
        A(0,2) =  0.00;
        A(1,0) =  0.00;
        A(1,1) =  0.00;
        A(1,2) = 12.50;
        A(2,0) = 13.75;
        A(2,1) =  0.00;
        A(2,2) = 14.00;
        A(3,0) =  0.00;
        A(3,1) = 16.50;
        A(3,2) =  0.00;

        {
            RowMajor    At;
            cusp::transpose(A, At);
            verify_result(At);
        }
        {
            ColumnMajor At;
            cusp::transpose(A, At);
            verify_result(At);
        }
        {
            RowMajor    At;
            At.resize(3,4,5);
            cusp::transpose(A, At);
            verify_result(At);
            ASSERT_EQUAL(At.pitch, 5);
        }
        {
            ColumnMajor At;
            At.resize(3,4,5);
            cusp::transpose(A, At);
            verify_result(At);
            ASSERT_EQUAL(At.pitch, 5);
        }
    }
    {
        ColumnMajor A(4,3);
        A.resize(4,3,5);
        A(0,0) = 10.25;
        A(0,1) = 11.00;
        A(0,2) =  0.00;
        A(1,0) =  0.00;
        A(1,1) =  0.00;
        A(1,2) = 12.50;
        A(2,0) = 13.75;
        A(2,1) =  0.00;
        A(2,2) = 14.00;
        A(3,0) =  0.00;
        A(3,1) = 16.50;
        A(3,2) =  0.00;

        {
            RowMajor    At;
            cusp::transpose(A, At);
            verify_result(At);
        }
        {
            ColumnMajor At;
            cusp::transpose(A, At);
            verify_result(At);
        }
        {
            RowMajor    At;
            At.resize(3,4,5);
            cusp::transpose(A, At);
            verify_result(At);
            ASSERT_EQUAL(At.pitch, 5);
        }
        {
            ColumnMajor At;
            At.resize(3,4,5);
            cusp::transpose(A, At);
            verify_result(At);
            ASSERT_EQUAL(At.pitch, 5);
        }
    }
}
DECLARE_HOST_DEVICE_UNITTEST(TestTransposeArray2dVariablePitch);

template <class Matrix>
void TestTranspose(void)
{
    TestTranspose<Matrix, Matrix>();
}
DECLARE_MATRIX_UNITTEST(TestTranspose);

