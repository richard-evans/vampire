#include <unittest/unittest.h>

#include <cusp/gallery/poisson.h>
#include <cusp/csr_matrix.h>
#include <cusp/krylov/cg_m.h>

template <class LinearOperator, class VectorType1, class VectorType2, class VectorType3>
void check_residuals(LinearOperator& A, VectorType1& xs, VectorType2& b, VectorType3& sigma)
{
    typedef typename LinearOperator::value_type   ValueType;
    typedef typename LinearOperator::memory_space MemorySpace;

    size_t N = A.num_rows;

    for (size_t i = 0; i < sigma.size(); i++)
    {
        // compute residual = b - (A + \sigma * I) x
        ValueType s = sigma[i];

        cusp::array1d<ValueType, MemorySpace> residual(A.num_rows, 0.0f);

        // TODO replace this with a array1d view of a array2d
        cusp::array1d<ValueType, MemorySpace> x(xs.begin() + i * N, xs.begin() + (i + 1) * N);
        cusp::multiply(A, x, residual);
        cusp::blas::axpby(residual, x, residual,  1.0f,     s);
        cusp::blas::axpby(residual, b, residual, -1.0f,  1.0f);

        ASSERT_EQUAL(cusp::blas::nrm2(residual) < 1e-4 * cusp::blas::nrm2(b), true);

        //std::cout << "Residual for sigma = " << s << " is " << cusp::blas::nrm2(residual) << std::endl;
    }
} // end check_residuals


template <class MemorySpace>
void TestConjugateGradientM(void)
{
    // which floating point type to use
    typedef float ValueType;

    // create an empty sparse matrix structure (HYB format)
    cusp::csr_matrix<int, ValueType, MemorySpace> A;

    // create a 2d Poisson problem on a 10x10 mesh
    cusp::gallery::poisson5pt(A, 10, 10);

    // allocate storage for solution (x) and right hand side (b)
    size_t N_s = 4;
    cusp::array1d<ValueType, MemorySpace> x(A.num_rows*N_s, ValueType(0));
    cusp::array1d<ValueType, MemorySpace> b(A.num_rows, ValueType(1));

    // set sigma values
    cusp::array1d<ValueType, MemorySpace> sigma(N_s);
    sigma[0] = ValueType(0.1);
    sigma[1] = ValueType(0.5);
    sigma[2] = ValueType(1.0);
    sigma[3] = ValueType(5.0);

    // set stopping criteria:
    //  iteration_limit    = 100
    //  relative_tolerance = 1e-6
    cusp::monitor<ValueType> monitor(b, 100, 1e-6);

    // solve the linear systems (A + \sigma_i * I) * x = b for each
    // sigma_i with the Conjugate Gradient method
    cusp::krylov::cg_m(A, x, b, sigma, monitor);

    check_residuals(A, x, b, sigma);
}
DECLARE_HOST_DEVICE_UNITTEST(TestConjugateGradientM);

