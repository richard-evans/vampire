#include <unittest/unittest.h>

#include <cusp/array1d.h>

#include <vector>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/zip_iterator.h>

template <typename MemorySpace>
void TestArray1dView(void)
{
    typedef typename cusp::array1d<int, MemorySpace> Array;
    typedef typename Array::iterator                 Iterator;
    typedef typename Array::const_iterator           ConstIterator;
    typedef typename std::vector<int>                Vector;
    typedef typename thrust::host_vector<int>        HostVector;
    typedef typename thrust::device_vector<int>      DeviceVector;

    // view to container
    {
        typedef cusp::array1d_view<Iterator> View;

        Array A(4);
        A[0] = 10;
        A[1] = 20;
        A[2] = 30;
        A[3] = 40;

        View V(A.begin(), A.end());

        ASSERT_EQUAL(V.size(),     4);
        ASSERT_EQUAL(V.capacity(), 4);
        ASSERT_EQUAL(V[0], 10);
        ASSERT_EQUAL(V[1], 20);
        ASSERT_EQUAL(V[2], 30);
        ASSERT_EQUAL(V[3], 40);
        ASSERT_EQUAL_QUIET(V.begin(), A.begin());
        ASSERT_EQUAL_QUIET(V.end(),   A.end());

        V[1] = 17;

        ASSERT_EQUAL(V[1], 17);
        ASSERT_EQUAL(A[1], 17);

        View W(A.begin(), A.end());
        ASSERT_EQUAL(W.size(),     4);
        ASSERT_EQUAL(W.capacity(), 4);
        ASSERT_EQUAL_QUIET(W.begin(), A.begin());
        ASSERT_EQUAL_QUIET(W.end(),   A.end());
    }

    // view to const container
    {
        typedef cusp::array1d_view<ConstIterator> View;

        const Array A(4, 10);

        View V(A.begin(), A.end());

        ASSERT_EQUAL(V.size(),     4);
        ASSERT_EQUAL(V.capacity(), 4);
        ASSERT_EQUAL(V[0], 10);
        ASSERT_EQUAL(V[1], 10);
        ASSERT_EQUAL(V[2], 10);
        ASSERT_EQUAL(V[3], 10);
        ASSERT_EQUAL_QUIET(V.begin(), A.begin());
        ASSERT_EQUAL_QUIET(V.end(),   A.end());

        View W(A.begin(), A.end());
        ASSERT_EQUAL(W.size(),     4);
        ASSERT_EQUAL(W.capacity(), 4);
        ASSERT_EQUAL_QUIET(W.begin(), A.begin());
        ASSERT_EQUAL_QUIET(W.end(),   A.end());
    }

    // const view to container
    {
        typedef const cusp::array1d_view<Iterator> View;

        Array A(4);
        A[0] = 10;
        A[1] = 20;
        A[2] = 30;
        A[3] = 40;

        View V(A);

        ASSERT_EQUAL(V.size(),     4);
        ASSERT_EQUAL(V.capacity(), 4);
        ASSERT_EQUAL(V[0], 10);
        ASSERT_EQUAL(V[1], 20);
        ASSERT_EQUAL(V[2], 30);
        ASSERT_EQUAL(V[3], 40);
        ASSERT_EQUAL_QUIET(V.begin(), A.begin());
        ASSERT_EQUAL_QUIET(V.end(),   A.end());

        V[1] = 17;

        ASSERT_EQUAL(V[1], 17);
        ASSERT_EQUAL(A[1], 17);

        View W(A.begin(), A.end());
        ASSERT_EQUAL(W.size(),     4);
        ASSERT_EQUAL(W.capacity(), 4);
        ASSERT_EQUAL_QUIET(W.begin(), A.begin());
        ASSERT_EQUAL_QUIET(W.end(),   A.end());
    }

    // const view to const container
    {
        typedef const cusp::array1d_view<ConstIterator> View;

        const Array A(4, 10);

        View V(A);

        ASSERT_EQUAL(V.size(),     4);
        ASSERT_EQUAL(V.capacity(), 4);
        ASSERT_EQUAL(V[0], 10);
        ASSERT_EQUAL(V[1], 10);
        ASSERT_EQUAL(V[2], 10);
        ASSERT_EQUAL(V[3], 10);
        ASSERT_EQUAL_QUIET(V.begin(), A.begin());
        ASSERT_EQUAL_QUIET(V.end(),   A.end());

        View W(A.begin(), A.end());
        ASSERT_EQUAL(W.size(),     4);
        ASSERT_EQUAL(W.capacity(), 4);
        ASSERT_EQUAL_QUIET(W.begin(), A.begin());
        ASSERT_EQUAL_QUIET(W.end(),   A.end());
    }


    // view to std::vector
    {
        typedef cusp::array1d_view<Vector::iterator> View;

        Vector A(4);

        View V(A);

        ASSERT_EQUAL(V.size(),     4);
        ASSERT_EQUAL(V.capacity(), 4);
        ASSERT_EQUAL_QUIET(V.begin(), A.begin());
        ASSERT_EQUAL_QUIET(V.end(),   A.end());

        View W(A.begin(), A.end());

        ASSERT_EQUAL(W.size(),     4);
        ASSERT_EQUAL(W.capacity(), 4);
        ASSERT_EQUAL_QUIET(W.begin(), A.begin());
        ASSERT_EQUAL_QUIET(W.end(),   A.end());

        View U = View(A);

        ASSERT_EQUAL(U.size(),     4);
        ASSERT_EQUAL(U.capacity(), 4);
        ASSERT_EQUAL_QUIET(U.begin(), A.begin());
        ASSERT_EQUAL_QUIET(U.end(),   A.end());
    }

    // view to thrust::host_vector
    {
        typedef cusp::array1d_view<HostVector::iterator> View;

        HostVector A(4);

        View V(A);

        ASSERT_EQUAL(V.size(),     4);
        ASSERT_EQUAL(V.capacity(), 4);
        ASSERT_EQUAL_QUIET(V.begin(), A.begin());
        ASSERT_EQUAL_QUIET(V.end(),   A.end());

        View W(A.begin(), A.end());

        ASSERT_EQUAL(W.size(),     4);
        ASSERT_EQUAL(W.capacity(), 4);
        ASSERT_EQUAL_QUIET(W.begin(), A.begin());
        ASSERT_EQUAL_QUIET(W.end(),   A.end());

        View U = View(A);

        ASSERT_EQUAL(U.size(),     4);
        ASSERT_EQUAL(U.capacity(), 4);
        ASSERT_EQUAL_QUIET(U.begin(), A.begin());
        ASSERT_EQUAL_QUIET(U.end(),   A.end());
    }

    // view to thrust::device_vector
    {
        typedef cusp::array1d_view<DeviceVector::iterator> View;

        DeviceVector A(4);

        View V(A);

        ASSERT_EQUAL(V.size(),     4);
        ASSERT_EQUAL(V.capacity(), 4);
        ASSERT_EQUAL_QUIET(V.begin(), A.begin());
        ASSERT_EQUAL_QUIET(V.end(),   A.end());

        View W(A.begin(), A.end());

        ASSERT_EQUAL(W.size(),     4);
        ASSERT_EQUAL(W.capacity(), 4);
        ASSERT_EQUAL_QUIET(W.begin(), A.begin());
        ASSERT_EQUAL_QUIET(W.end(),   A.end());

        View U = View(A);

        ASSERT_EQUAL(U.size(),     4);
        ASSERT_EQUAL(U.capacity(), 4);
        ASSERT_EQUAL_QUIET(U.begin(), A.begin());
        ASSERT_EQUAL_QUIET(U.end(),   A.end());
    }

}
DECLARE_HOST_DEVICE_UNITTEST(TestArray1dView);


template <typename MemorySpace>
void TestMakeArray1dView(void)
{
    typedef typename cusp::array1d<int, MemorySpace> Array;
    typedef typename Array::iterator                 Iterator;
    typedef cusp::array1d_view<Iterator>             View;
    typedef const Array                              ConstArray;
    typedef typename Array::const_iterator           ConstIterator;
    typedef cusp::array1d_view<ConstIterator>        ConstView;

    // construct from iterators
    {
        Array A(4);

        View V = cusp::make_array1d_view(A.begin(), A.end());

        ASSERT_EQUAL(V.size(),     4);
        ASSERT_EQUAL(V.capacity(), 4);
        ASSERT_EQUAL_QUIET(V.begin(), A.begin());
        ASSERT_EQUAL_QUIET(V.end(),   A.end());

        // check that view::iterator is mutable
        V[1] = 17;

        ASSERT_EQUAL(V[1], 17);
        ASSERT_EQUAL(A[1], 17);
    }

    // construct from container
    {
        Array A(4);

        View V = cusp::make_array1d_view(A);

        ASSERT_EQUAL(V.size(),     4);
        ASSERT_EQUAL(V.capacity(), 4);
        ASSERT_EQUAL_QUIET(V.begin(), A.begin());
        ASSERT_EQUAL_QUIET(V.end(),   A.end());

        // check that view::iterator is mutable
        V[1] = 17;

        ASSERT_EQUAL(V[1], 17);
        ASSERT_EQUAL(A[1], 17);
    }

    // construct from const container
    {
        ConstArray A(4);

        ConstView V = cusp::make_array1d_view(A);

        ASSERT_EQUAL(V.size(),     4);
        ASSERT_EQUAL(V.capacity(), 4);
        ASSERT_EQUAL_QUIET(V.begin(), A.begin());
        ASSERT_EQUAL_QUIET(V.end(),   A.end());
    }

    // construct from view
    {
        Array A(4);

        View X = cusp::make_array1d_view(A);
        View V = cusp::make_array1d_view(X);

        ASSERT_EQUAL(V.size(),     4);
        ASSERT_EQUAL(V.capacity(), 4);
        ASSERT_EQUAL_QUIET(V.begin(), A.begin());
        ASSERT_EQUAL_QUIET(V.end(),   A.end());

        // check that view::iterator is mutable
        V[1] = 17;

        ASSERT_EQUAL(V[1], 17);
        ASSERT_EQUAL(A[1], 17);
    }
}
DECLARE_HOST_DEVICE_UNITTEST(TestMakeArray1dView);


template <typename MemorySpace>
void TestArray1dViewAssignment(void)
{
    typedef typename cusp::array1d<int, MemorySpace> Array;
    typedef typename Array::iterator                 Iterator;
    typedef cusp::array1d_view<Iterator>             View;

    Array A(4);
    Array B(8);

    View V(A.begin(), A.end());

    ASSERT_EQUAL(V.size(),     4);
    ASSERT_EQUAL(V.capacity(), 4);
    ASSERT_EQUAL_QUIET(V.begin(), A.begin());
    ASSERT_EQUAL_QUIET(V.end(),   A.end());

    V = View(A);

    ASSERT_EQUAL(V.size(),     4);
    ASSERT_EQUAL(V.capacity(), 4);
    ASSERT_EQUAL_QUIET(V.begin(), A.begin());
    ASSERT_EQUAL_QUIET(V.end(),   A.end());

    V = View(B);

    ASSERT_EQUAL(V.size(),     8);
    ASSERT_EQUAL(V.capacity(), 8);
    ASSERT_EQUAL_QUIET(V.begin(), B.begin());
    ASSERT_EQUAL_QUIET(V.end(),   B.end());

    const View W = View(V);

    ASSERT_EQUAL(W.size(),     8);
    ASSERT_EQUAL(W.capacity(), 8);
    ASSERT_EQUAL_QUIET(W.begin(), B.begin());
    ASSERT_EQUAL_QUIET(W.end(),   B.end());
}
DECLARE_HOST_DEVICE_UNITTEST(TestArray1dViewAssignment);

template <typename MemorySpace>
void TestArray1dViewResize(void)
{
    typedef typename cusp::array1d<int, MemorySpace> Array;
    typedef typename Array::iterator                 Iterator;
    typedef cusp::array1d_view<Iterator>             View;

    Array A(4);
    A[0] = 10;
    A[1] = 20;
    A[2] = 30;
    A[3] = 40;

    View V(A.begin(), A.end());

    V.resize(3);

    ASSERT_EQUAL(V.size(),     3);
    ASSERT_EQUAL(V.capacity(), 4);
    ASSERT_EQUAL_QUIET(V.begin(), A.begin());
    ASSERT_EQUAL_QUIET(V.end(),   A.begin() + 3);

    V.resize(2);

    ASSERT_EQUAL(V.size(),     2);
    ASSERT_EQUAL(V.capacity(), 4);
    ASSERT_EQUAL_QUIET(V.begin(), A.begin());
    ASSERT_EQUAL_QUIET(V.end(),   A.begin() + 2);

    V.resize(4);

    ASSERT_EQUAL(V.size(),     4);
    ASSERT_EQUAL(V.capacity(), 4);
    ASSERT_EQUAL_QUIET(V.begin(), A.begin());
    ASSERT_EQUAL_QUIET(V.end(),   A.begin() + 4);

    ASSERT_THROWS(V.resize(5), cusp::not_implemented_exception);
}
DECLARE_HOST_DEVICE_UNITTEST(TestArray1dViewResize);


template <typename MemorySpace>
void TestArray1dViewSize(void)
{
    typedef typename cusp::array1d<int, MemorySpace> Array;
    typedef typename Array::iterator                 Iterator;
    typedef cusp::array1d_view<Iterator>             View;

    Array A(4);

    View V(A.begin(), A.end());

    ASSERT_EQUAL(V.size(), 4);

    V.resize(2);

    ASSERT_EQUAL(V.size(), 2);

    View W = V;

    ASSERT_EQUAL(W.size(), 2);

    V = View(A);

    ASSERT_EQUAL(V.size(), 4);
}
DECLARE_HOST_DEVICE_UNITTEST(TestArray1dViewSize);


template <typename MemorySpace>
void TestArray1dViewCapacity(void)
{
    typedef typename cusp::array1d<int, MemorySpace> Array;
    typedef typename Array::iterator                 Iterator;
    typedef cusp::array1d_view<Iterator>             View;

    Array A(4);

    View V(A.begin(), A.end());

    ASSERT_EQUAL(V.size(),     4);
    ASSERT_EQUAL(V.capacity(), 4);

    A.resize(2);

    V = View(A);

    ASSERT_EQUAL(V.size(),     2);
    ASSERT_EQUAL(V.capacity(), 4);
}
DECLARE_HOST_DEVICE_UNITTEST(TestArray1dViewCapacity);


template <typename MemorySpace>
void TestArray1dViewCountingIterator(void)
{
    typedef thrust::counting_iterator<int> Iterator;

    cusp::array1d_view<Iterator> V(Iterator(5), Iterator(9));

    ASSERT_EQUAL(V.size(), 4);
    ASSERT_EQUAL(V[0], 5);
    ASSERT_EQUAL(V[3], 8);

    cusp::counting_array<int> W(4, 5);
    ASSERT_EQUAL(W.size(), 4);
    ASSERT_EQUAL(W[0], 5);
    ASSERT_EQUAL(W[3], 8);

    cusp::constant_array<int> X(200, 5);
    ASSERT_EQUAL(X[0], 5);
    ASSERT_EQUAL(X[3], 5);
    ASSERT_EQUAL(X[199],5);
}
DECLARE_HOST_DEVICE_UNITTEST(TestArray1dViewCountingIterator);

template <typename MemorySpace>
void TestArray1dViewZipIterator(void)
{
    cusp::array1d<int, MemorySpace> A(4);
    cusp::array1d<int, MemorySpace> B(4);
    A[0] = 10;
    A[1] = 20;
    A[2] = 30;
    A[3] = 40;
    B[0] = 50;
    B[1] = 60;
    B[2] = 70;
    B[3] = 80;

    typedef typename cusp::array1d<int, MemorySpace>::iterator Iterator;
    typedef typename thrust::tuple<Iterator,Iterator>          IteratorTuple;
    typedef typename thrust::zip_iterator<IteratorTuple>       ZipIterator;

    ZipIterator begin = thrust::make_zip_iterator(thrust::make_tuple(A.begin(), B.begin()));

    cusp::array1d_view<ZipIterator> V(begin, begin + 4);

    ASSERT_EQUAL(V.size(), 4);
    ASSERT_EQUAL_QUIET(V[0], thrust::make_tuple(10,50));
    ASSERT_EQUAL_QUIET(V[3], thrust::make_tuple(40,80));
}
DECLARE_HOST_DEVICE_UNITTEST(TestArray1dViewZipIterator);


template <typename MemorySpace>
void TestArray1dViewEquality(void)
{
    typedef typename cusp::array1d<int, MemorySpace> Array;
    typedef typename Array::iterator                 Iterator;
    typedef cusp::array1d_view<Iterator>             View;

    Array A(2);
    A[0] = 10;
    A[1] = 20;

    Array B(3);
    B[0] = 10;
    B[1] = 20;
    B[2] = 30;

    View V(A);
    View W(B);

    ASSERT_EQUAL_QUIET(A == V, true);
    ASSERT_EQUAL_QUIET(V == A, true);
    ASSERT_EQUAL_QUIET(V == V, true);
    ASSERT_EQUAL_QUIET(A != V, false);
    ASSERT_EQUAL_QUIET(V != A, false);
    ASSERT_EQUAL_QUIET(V != V, false);

    ASSERT_EQUAL_QUIET(V == B, false);
    ASSERT_EQUAL_QUIET(B == V, false);
    ASSERT_EQUAL_QUIET(V == W, false);
    ASSERT_EQUAL_QUIET(W == V, false);
    ASSERT_EQUAL_QUIET(V != B, true);
    ASSERT_EQUAL_QUIET(B != V, true);
    ASSERT_EQUAL_QUIET(V != W, true);
    ASSERT_EQUAL_QUIET(W != V, true);

    W.resize(2);

    ASSERT_EQUAL_QUIET(V == W, true);
    ASSERT_EQUAL_QUIET(V != W, false);
}
DECLARE_HOST_DEVICE_UNITTEST(TestArray1dViewEquality);

template <typename MemorySpace>
void TestArray1dViewSubarray(void)
{
    typedef typename cusp::array1d<int, MemorySpace> Array;
    typedef typename Array::iterator                 Iterator;
    typedef cusp::array1d_view<Iterator>             View;

    Array A(4);
    A[0] = 10;
    A[1] = 20;
    A[2] = 30;
    A[3] = 40;

    View V = A.subarray(1,3);

    ASSERT_EQUAL(V.size(),     3);
    ASSERT_EQUAL_QUIET(V.begin(), A.begin() + 1);
    ASSERT_EQUAL_QUIET(V.end(),   A.begin() + 4);

    V = A.subarray(0,1);

    ASSERT_EQUAL(V.size(),     1);
    ASSERT_EQUAL_QUIET(V.begin(), A.begin() + 0);
    ASSERT_EQUAL_QUIET(V.end(),   A.begin() + 1);

    V = A.subarray(1,3);
    View W = V.subarray(0,1);

    ASSERT_EQUAL(W.size(),     1);
    ASSERT_EQUAL_QUIET(W.begin(), A.begin() + 1);
    ASSERT_EQUAL_QUIET(W.end(),   A.begin() + 2);
}
DECLARE_HOST_DEVICE_UNITTEST(TestArray1dViewSubarray);
