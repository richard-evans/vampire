//-----------------------------------------------------------------------------
//
// This source file is part of the VAMPIRE open source package under the
// GNU GPL (version 2) licence (see licence file for details).
//
// (c) R F L Evans 2015. All rights reserved.
//
//-----------------------------------------------------------------------------

// C++ standard library headers

// Vampire headers
#include "cuda.hpp"

// Local cuda headers
#include "cuda_utils.hpp"
#include "internal.hpp"
#include "statistics.hpp"
#include "typedefs.hpp"

#ifdef CUDA
namespace cu = vcuda::internal;
#endif

namespace vcuda{

#ifdef CUDA

   namespace stats
   {
         void update ()
         {

            // If enabled use CPU to calculate statistics by copying data from GPU
            if(vcuda::internal::stats::use_cpu){

				   // copy spin data to CPU
               /*
               thrust::copy(internal::atoms::x_spin_array.begin(),internal::atoms::x_spin_array.end(),::atoms::x_spin_array.begin());
               thrust::copy(internal::atoms::y_spin_array.begin(),internal::atoms::y_spin_array.end(),::atoms::y_spin_array.begin());
               thrust::copy(internal::atoms::z_spin_array.begin(),internal::atoms::z_spin_array.end(),::atoms::z_spin_array.begin());
               */
               hipMemcpy(::atoms::x_spin_array.data(), internal::atoms::d_x_spin, ::atoms::num_atoms * sizeof(cu::cu_real_t), hipMemcpyDeviceToHost);
               hipMemcpy(::atoms::y_spin_array.data(), internal::atoms::d_y_spin, ::atoms::num_atoms * sizeof(cu::cu_real_t), hipMemcpyDeviceToHost);
               hipMemcpy(::atoms::z_spin_array.data(), internal::atoms::d_z_spin, ::atoms::num_atoms * sizeof(cu::cu_real_t), hipMemcpyDeviceToHost);

               // call cpu statistics functions
               if(::stats::calculate_system_magnetization)          ::stats::system_magnetization.calculate_magnetization(::atoms::x_spin_array, ::atoms::y_spin_array, ::atoms::z_spin_array, ::atoms::m_spin_array);
               if(::stats::calculate_material_magnetization)        ::stats::material_magnetization.calculate_magnetization(::atoms::x_spin_array, ::atoms::y_spin_array, ::atoms::z_spin_array, ::atoms::m_spin_array);
               if(::stats::calculate_height_magnetization)          ::stats::height_magnetization.calculate_magnetization(::atoms::x_spin_array, ::atoms::y_spin_array, ::atoms::z_spin_array, ::atoms::m_spin_array);
               if(::stats::calculate_material_height_magnetization) ::stats::material_height_magnetization.calculate_magnetization(::atoms::x_spin_array, ::atoms::y_spin_array, ::atoms::z_spin_array, ::atoms::m_spin_array);

               // return before doing the GPU version
               return;
            }


            // increase the counter
            cu::stats::counter++;

         }

         void get ()
         {

            // If CPU stats calculation do nothing
            if(vcuda::internal::stats::use_cpu) return;
         }

         void reset ()
         {
         }

   } /* stats */

   namespace internal
   {
      namespace stats
      {
      } /* stats */
   } /* internal */

#endif

} // end of namespace cuda
