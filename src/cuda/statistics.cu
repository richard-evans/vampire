#include "hip/hip_runtime.h"
//-----------------------------------------------------------------------------
//
// This source file is part of the VAMPIRE open source package under the
// GNU GPL (version 2) licence (see licence file for details).
//
// (c) R F L Evans 2015. All rights reserved.
//
//-----------------------------------------------------------------------------

// C++ standard library headers

// Vampire headers
#include "cuda.hpp"

// Local cuda headers
#include "cuda_utils.hpp"
#include "internal.hpp"
#include "statistics.hpp"

#ifdef CUDA
namespace cu = vcuda::internal;
#endif

namespace vcuda{

#ifdef CUDA

   namespace stats
   {
         void update ()
         {

            // If enabled use CPU to calculate statistics by copying data from GPU
            if(vcuda::internal::stats::use_cpu){

				   // copy spin data to CPU
               thrust::copy(internal::atoms::x_spin_array.begin(),internal::atoms::x_spin_array.end(),::atoms::x_spin_array.begin());
               thrust::copy(internal::atoms::y_spin_array.begin(),internal::atoms::y_spin_array.end(),::atoms::y_spin_array.begin());
               thrust::copy(internal::atoms::z_spin_array.begin(),internal::atoms::z_spin_array.end(),::atoms::z_spin_array.begin());

               // call cpu statistics functions
               if(::stats::calculate_system_magnetization)          ::stats::system_magnetization.calculate_magnetization(::atoms::x_spin_array, ::atoms::y_spin_array, ::atoms::z_spin_array, ::atoms::m_spin_array);
               if(::stats::calculate_material_magnetization)        ::stats::material_magnetization.calculate_magnetization(::atoms::x_spin_array, ::atoms::y_spin_array, ::atoms::z_spin_array, ::atoms::m_spin_array);
               if(::stats::calculate_height_magnetization)          ::stats::height_magnetization.calculate_magnetization(::atoms::x_spin_array, ::atoms::y_spin_array, ::atoms::z_spin_array, ::atoms::m_spin_array);
               if(::stats::calculate_material_height_magnetization) ::stats::material_height_magnetization.calculate_magnetization(::atoms::x_spin_array, ::atoms::y_spin_array, ::atoms::z_spin_array, ::atoms::m_spin_array);

               // return before doing the GPU version
               return;
            }

            cu::stats::__update_stat (
                  cu::stats::system_mask,
                  cu::stats::system_magnetization,
                  cu::stats::system_mean_magnetization,
                  cu::stats::system_mask_size);

            cu::stats::__update_stat (
                  cu::stats::material_mask,
                  cu::stats::material_magnetization,
                  cu::stats::material_mean_magnetization,
                  cu::stats::material_mask_size);

            cu::stats::__update_stat (
                  cu::stats::height_mask,
                  cu::stats::height_magnetization,
                  cu::stats::height_mean_magnetization,
                  cu::stats::height_mask_size);

            cu::stats::__update_stat (
                  cu::stats::material_height_mask,
                  cu::stats::material_height_magnetization,
                  cu::stats::material_height_mean_magnetization,
                  cu::stats::material_height_mask_size);

            // increase the counter
            cu::stats::counter++;

         }

         void get ()
         {

            // If CPU stats calculation do nothing
            if(vcuda::internal::stats::use_cpu) return;

            cu::stats::__get_stat (
                  cu::stats::system_magnetization,
                  cu::stats::system_mean_magnetization,
                  ::stats::system_magnetization
                  );

            cu::stats::__get_stat (
                  cu::stats::material_magnetization,
                  cu::stats::material_mean_magnetization,
                  ::stats::material_magnetization
                  );

            cu::stats::__get_stat (
                  cu::stats::height_magnetization,
                  cu::stats::height_mean_magnetization,
                  ::stats::height_magnetization
                  );

            cu::stats::__get_stat (
                  cu::stats::material_height_magnetization,
                  cu::stats::material_height_mean_magnetization,
                  ::stats::material_height_magnetization
                  );
         }

         void reset ()
         {

            // reset magnetization statistics
            if(vcuda::internal::stats::use_cpu){
               if(::stats::calculate_system_magnetization)          ::stats::system_magnetization.reset_magnetization_averages();
               if(::stats::calculate_material_magnetization)        ::stats::material_magnetization.reset_magnetization_averages();
               if(::stats::calculate_height_magnetization)          ::stats::height_magnetization.reset_magnetization_averages();
               if(::stats::calculate_material_height_magnetization) ::stats::material_height_magnetization.reset_magnetization_averages();
               return;
            }

            cu::stats::counter = 0L;

            cu::stats::__reset_stat (
                  cu::stats::system_magnetization,
                  cu::stats::system_mean_magnetization
                  );

            cu::stats::__reset_stat (
                  cu::stats::material_magnetization,
                  cu::stats::material_mean_magnetization
                  );

            cu::stats::__reset_stat (
                  cu::stats::height_magnetization,
                  cu::stats::height_mean_magnetization
                  );

            cu::stats::__reset_stat (
                  cu::stats::material_height_magnetization,
                  cu::stats::material_height_mean_magnetization
                  );
         }

   } /* stats */

   namespace internal
   {
      namespace stats
      {


         void __update_stat (
               const cu_index_array_t & mask,
               cu_real_array_t & stat,
               cu_real_array_t & mean_stat,
               int mask_size
               )
         {

            if (mask_size < 1) return; // Nothing to do

            // Clean up the stat buffer
            thrust::fill(
                  stat.begin(),
                  stat.end(),
                  0.0);

            const int * d_mask = thrust::raw_pointer_cast (
                  mask.data());
            cu_real_t * d_stat = thrust::raw_pointer_cast (
                  stat.data());
            cu_real_t * d_accu = thrust::raw_pointer_cast (
                  mean_stat.data());

            cu_real_t * d_x_spin = thrust::raw_pointer_cast(
                  cu::atoms::x_spin_array.data());
            cu_real_t * d_y_spin = thrust::raw_pointer_cast(
                  cu::atoms::y_spin_array.data());
            cu_real_t * d_z_spin = thrust::raw_pointer_cast(
                  cu::atoms::z_spin_array.data());
            cu_real_t * d_spin_norm = thrust::raw_pointer_cast(
                  cu::atoms::spin_norm_array.data());

            int n_bins = mask_size;
            int n_atoms = mask.size ();

            if (n_bins < 8) {
               hist_by_key_smaller_mask <<< n_bins * 4, 512 >>> (
                     d_x_spin,
                     d_y_spin,
                     d_z_spin,
                     d_spin_norm,
                     d_mask,
                     d_stat,
                     n_bins,
                     n_atoms
                     );
               check_cuda_errors (__FILE__, __LINE__);
            }
            else if (n_bins < 128)
            {

                // Use the shared memory implementation

               int n_bytes = 4 * mask_size * sizeof(cu_real_array_t::value_type);
               hist_by_key_small_mask <<< cu::grid_size, cu::block_size, n_bytes >>> (
                     d_x_spin,
                     d_y_spin,
                     d_z_spin,
                     d_spin_norm,
                     d_mask,
                     d_stat,
                     n_bins,
                     n_atoms
                     );
               check_cuda_errors (__FILE__, __LINE__);
            }
            else
            {

               // Use the brute force implementation

               hist_by_key_big_mask <<< cu::grid_size, cu::block_size >>> (
                     d_x_spin,
                     d_y_spin,
                     d_z_spin,
                     d_spin_norm,
                     d_mask,
                     d_stat,
                     n_bins,
                     n_atoms
                     );
               check_cuda_errors (__FILE__, __LINE__);
            }

             // Reduce and accumulate

            int gs = n_bins / cu::block_size + 1;
            update_norm_and_accum <<< gs , cu::block_size >>> (
                  d_stat,
                  d_accu,
                  n_bins
                  );

            check_cuda_errors (__FILE__, __LINE__);

         }


         void __get_stat (
               const cu_real_array_t& stat,
               const cu_real_array_t& mean_stat,
               ::stats::magnetization_statistic_t& local_stat
               )
         {

            /*
             * Copy to local arrays
             */

            thrust::host_vector<cu_real_t> h_stat(stat.size());
            thrust::host_vector<cu_real_t> h_mean_stat(mean_stat.size());

            thrust::copy(stat.begin(), stat.end(), h_stat.begin());
            thrust::copy(mean_stat.begin(), mean_stat.end(), h_mean_stat.begin());

            /*
             * Call the method in the magnetization_statistic_t instance
             */

            std::vector<double> stl_stat (h_stat.begin(), h_stat.end());
            std::vector<double> stl_mean_stat (h_mean_stat.begin(), h_mean_stat.end());

            // We're keeping the averages in the GPU
            local_stat.reset_magnetization_averages();
            local_stat.set_magnetization (
                  stl_stat,
                  stl_mean_stat,
                  cu::stats::counter);
            check_cuda_errors (__FILE__, __LINE__);

         }


         void __reset_stat (
               cu_real_array_t& stat,
               cu_real_array_t& mean_stat
               )
         {
            thrust::fill(
                  stat.begin(),
                  stat.end(),
                  0.0);
            thrust::fill(
                  mean_stat.begin(),
                  mean_stat.end(),
                  0.0);
            check_cuda_errors (__FILE__, __LINE__);
         }

         __global__ void hist_by_key_smaller_mask (
               const cu_real_t * __restrict__ x_spin,
               const cu_real_t * __restrict__ y_spin,
               const cu_real_t * __restrict__ z_spin,
               const cu_real_t * __restrict__ norm_spin,
               const int * __restrict__ mask,
               cu_real_t * hist,
               int n_bins,
               int n_atoms
               )
         {
            cu_real_t val = 0; // Initialize the register

            int rol = blockIdx.x % 4;
            int bin = blockIdx.x / 4;
            // Block stride loop (each block reads the whole array)
            for ( int i = threadIdx.x;
                  i < n_atoms;
                  i += blockDim.x)
            {
               // Single spin test
               // if (i != 0) continue;
               cu_real_t mu_s = norm_spin[i];
               if (rol == 0) val += mask[i] == bin ? x_spin[i] * mu_s : 0;
               if (rol == 1) val += mask[i] == bin ? y_spin[i] * mu_s : 0;
               if (rol == 2) val += mask[i] == bin ? z_spin[i] * mu_s : 0;
               if (rol == 3) val += mask[i] == bin ? mu_s : 0;
            }

            typedef hipcub::BlockReduce<cu_real_t, 512> BlockReduce;
            __shared__ typename BlockReduce::TempStorage temp_storage;
            val = BlockReduce(temp_storage).Sum(val);

            if (threadIdx.x == 0) {
               hist[4 * bin + rol] += val;
            }
         }

         /**
           * Naivive implementation of histogram using shared memory
           * FIXME: Prone to improvements
           */
         __global__ void hist_by_key_small_mask (
               const cu_real_t * __restrict__ x_spin,
               const cu_real_t * __restrict__ y_spin,
               const cu_real_t * __restrict__ z_spin,
               const cu_real_t * __restrict__ norm_spin,
               const int * __restrict__ mask,
               cu_real_t * hist,
               int n_bins,
               int n_atoms
               )
         {
            extern __shared__ cu_real_t block_hist[];

            for (int i = threadIdx.x; i < 4 * n_bins; i += blockDim.x)
            {
               /*
                * Initialize block memory
                */
               block_hist[i] = 0.0;
            }

            __syncthreads ();

            for ( int i = blockIdx.x * blockDim.x + threadIdx.x;
                  i < n_atoms;
                  i += blockDim.x * gridDim.x)
            {
               /*
                * Store stuff in the shared memory
                */
               int bin = mask[i];
               cu_real_t mu_s = norm_spin[i];
               cu::atomicAdd (block_hist + 4 * bin + 0, x_spin[i] * mu_s);
               cu::atomicAdd (block_hist + 4 * bin + 1, y_spin[i] * mu_s);
               cu::atomicAdd (block_hist + 4 * bin + 2, z_spin[i] * mu_s);
               cu::atomicAdd (block_hist + 4 * bin + 3, mu_s);
            }

            __syncthreads ();

            for (int i = threadIdx.x; i < 4 * n_bins; i += blockDim.x)
            {
               /*
                * Store stuff in the main memory
                */
               cu::atomicAdd (hist + 4 * i + 0, block_hist[4 * i + 0]);
               cu::atomicAdd (hist + 4 * i + 1, block_hist[4 * i + 1]);
               cu::atomicAdd (hist + 4 * i + 2, block_hist[4 * i + 2]);
               cu::atomicAdd (hist + 4 * i + 3, block_hist[4 * i + 3]);
            }

         }


         __global__ void hist_by_key_big_mask (
               const cu_real_t * __restrict__ x_spin,
               const cu_real_t * __restrict__ y_spin,
               const cu_real_t * __restrict__ z_spin,
               const cu_real_t * __restrict__ norm_spin,
               const int * __restrict__ mask,
               cu_real_t * hist,
               int n_bins,
               int n_atoms
               )
         {

            for ( int i = blockIdx.x * blockDim.x + threadIdx.x;
                  i < n_atoms;
                  i += blockDim.x * gridDim.x)
            {

                // Store stuff in the main memory

               int bin = mask[i];
               cu_real_t mu_s = norm_spin[i];
               cu::atomicAdd (hist + 4 * bin + 0, x_spin[i] * mu_s);
               cu::atomicAdd (hist + 4 * bin + 1, y_spin[i] * mu_s);
               cu::atomicAdd (hist + 4 * bin + 2, z_spin[i] * mu_s);
               cu::atomicAdd (hist + 4 * bin + 3, mu_s);
            }
         }


         __global__ void update_norm_and_accum (
               cu_real_t * hist,
               cu_real_t * accum,
               int n_bins
               )
         {

            for ( int i = blockIdx.x * blockDim.x + threadIdx.x;
                  i < n_bins;
                  i += blockDim.x * gridDim.x)
            {
               cu_real_t mx = hist[4 * i + 0];
               cu_real_t my = hist[4 * i + 1];
               cu_real_t mz = hist[4 * i + 2];
               cu_real_t ms = hist[4 * i + 3];

               if (ms > 0.0) {

                  // cuda overloads reciprocal sqrt for float and double
                  cu_real_t imm = rsqrt(mx * mx + my * my + mz * mz);

                  hist[4 * i + 0] = mx * imm;
                  hist[4 * i + 1] = my * imm;
                  hist[4 * i + 2] = mz * imm;
                  hist[4 * i + 3] = imm * ms;
                  accum[4 * i + 0] += mx * imm;
                  accum[4 * i + 1] += my * imm;
                  accum[4 * i + 2] += mz * imm;
                  accum[4 * i + 3] += imm * ms;
               } else {
                  // Just wipe the histogram
                  // FIXME Even this could be removed
                  for (int j = 0; j < 4; j++)
                     hist[4 * i + j] = 0.0;
               }
            }
         }

      } /* stats */
   } /* internal */

#endif

} // end of namespace cuda
