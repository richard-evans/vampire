#include "hip/hip_runtime.h"
//-----------------------------------------------------------------------------
//
// This source file is part of the VAMPIRE open source package under the
// GNU GPL (version 2) licence (see licence file for details).
//
// (c) R F L Evans 2015. All rights reserved.
//
//-----------------------------------------------------------------------------

// C++ standard library headers

#include <vector>

// Vampire headers
#include "atoms.hpp"
#include "cuda.hpp"
#include "errors.hpp"
#include "gpu.hpp"
#include "random.hpp"
#include "stats.hpp"
#include "vio.hpp"

// Local cuda headers

#include "cuda_utils.hpp"
#include "data.hpp"
#include "internal.hpp"

#include "exchange_fields.hpp"
#include "statistics.hpp"

#ifdef CUDA
namespace cu = ::vcuda::internal;
#endif

namespace vcuda{

   //-------------------------------------------------------------------------------
   // Function to initialize GPU data
   //-------------------------------------------------------------------------------
   bool initialize(bool cpu_stats){

#ifdef CUDA

      std::cout << "CUDA has been enabled in ";
      zlog << zTs() << "CUDA has been enabled in ";
      #ifdef CUDA_DP
         std::cout << "double precision mode" << std::endl;
         zlog << "double precision mode" << std::endl;
      #else
         std::cout << "single precision mode" << std::endl;
         zlog << "single precision mode" << std::endl;
      #endif

      // set internal cpu statistics flag
      vcuda::internal::stats::use_cpu = cpu_stats;

      // Check if there is a compatible devices
      int n_devices;
      hipError_t error = hipGetDeviceCount(&n_devices);

      if( error == hipErrorNoDevice )
      {
         std::cout << "Error: CUDA is enabled but no CUDA compatible devices are available." << std::endl;
         zlog << zTs() << "Error: CUDA is enabled but no CUDA compatible devices are available." << std::endl;
         ::err::vexit();
      }
      else if ( error == hipErrorInsufficientDriver )
      {
         std::cout     << "Error: CUDA is enabled but no CUDA drivers are incompatible. Please update drivers." << std::endl;
         zlog << zTs() << "Error: CUDA is enabled but no CUDA drivers are incompatible. Please update drivers." << std::endl;
         ::err::vexit();
      }
      else if ( error != hipSuccess)
      {
         std::cout     << "Error: CUDA is enabled but error querying devices." << std::endl;
         zlog << zTs() << "Error: CUDA is enabled but error querying devices." << std::endl;
         ::err::vexit();
      }

      // Set cuda device if specified by user
      if(gpu::device != -1 && gpu::device < n_devices){
         zlog << zTs() << "Setting CUDA device to " << gpu::device << std::endl;
         hipError_t error = hipSetDevice(gpu::device);
         if( error == hipErrorSetOnActiveProcess )
         {
            std::cerr     << "Error: CUDA is unable to set active process to device " << gpu::device << std::endl;
            zlog << zTs() << "Error: CUDA is unable to set active process to device " << gpu::device << std::endl;
            ::err::vexit();
         }
         else if ( error == hipErrorInvalidDevice )
         {
            std::cerr     << "Error: CUDA is requesting device " << gpu::device << " which is an invalid device." << std::endl;
            zlog << zTs() << "Error: CUDA is requesting device " << gpu::device << " which is an invalid device." << std::endl;
            ::err::vexit();
         }
         else if ( error != hipSuccess)
         {
            std::cerr     << "Error: CUDA unable to set device to " << gpu::device << std::endl;
            zlog << zTs() << "Error: CUDA unable to set device to " << gpu::device << std::endl;
            ::err::vexit();
         }
      }

      bool success = true;

      /*
       * Set the block_size according to the number of atoms
       */

      size_t _grid_size = ( (::atoms::num_atoms/2) / cu::block_size) + 1;

      //std::cerr << "Natoms = " << ::atoms::num_atoms << "\n";
      //std::cerr << "Block size = " << cu::block_size << "\n";
      //std::cerr << "grid1 = " << _grid_size << "\n";
      //std::cerr << "grid2 = " << cu::grid_size << "\n";

      // I dont think this does anything
      //if (_grid_size < cu::grid_size)
      //   cu::grid_size = _grid_size;

      cu::grid_size = _grid_size;
      //std::cerr << "grid2 = " << cu::grid_size << std::endl;

      zlog << zTs() << "Using cuda version with block size " << cu::block_size << " and grid size " << cu::grid_size << std::endl;

      success = success && cu::__initialize_atoms ();
      success = success && cu::__initialize_fields ();
      success = success && cu::__initialize_cells ();
      success = success && cu::__initialize_materials ();
      success = success && cu::__initialize_topology ();
      success = success && cu::__initialize_curand ();
      success = success && cu::__initialize_stats ();

      // Set up the exchange fields
      if( cu::exchange::initialise_exchange() != EXIT_SUCCESS)
      {
         std::cerr << "Failed to initialise exchange" << std::endl;
         success = false;
      }

      // Successful initialization
      return success;

#else
      // Default (initializtion failed)
      return false;
#endif
   }

#ifdef CUDA

   namespace internal {

      bool __initialize_atoms ()
      {
         /*
          * Allocate memory in the device and transfer the
          * spins of the atoms.
          */

         cu::atoms::x_spin_array.resize(::atoms::num_atoms);
         cu::atoms::y_spin_array.resize(::atoms::num_atoms);
         cu::atoms::z_spin_array.resize(::atoms::num_atoms);

         thrust::copy(
               ::atoms::x_spin_array.begin(),
               ::atoms::x_spin_array.end(),
               cu::atoms::x_spin_array.begin()
               );

         thrust::copy(
               ::atoms::y_spin_array.begin(),
               ::atoms::y_spin_array.end(),
               cu::atoms::y_spin_array.begin()
               );

         thrust::copy(
               ::atoms::z_spin_array.begin(),
               ::atoms::z_spin_array.end(),
               cu::atoms::z_spin_array.begin()
               );

         /*
          * Allocate memory in the device and transfer the
          * coordinates of the atoms.
          */

         cu::atoms::x_coord_array.resize(::atoms::num_atoms);
         cu::atoms::y_coord_array.resize(::atoms::num_atoms);
         cu::atoms::z_coord_array.resize(::atoms::num_atoms);

         thrust::copy(
               ::atoms::x_coord_array.begin(),
               ::atoms::x_coord_array.end(),
               cu::atoms::x_coord_array.begin()
               );

         thrust::copy(
               ::atoms::y_coord_array.begin(),
               ::atoms::y_coord_array.end(),
               cu::atoms::y_coord_array.begin()
               );

         thrust::copy(
               ::atoms::z_coord_array.begin(),
               ::atoms::z_coord_array.end(),
               cu::atoms::z_coord_array.begin()
               );

         /*
          * Allocate memory and send information about the types of
          * atoms
          */

         cu::atoms::type_array.resize(::atoms::num_atoms);

         thrust::copy(
               ::atoms::type_array.begin(),
               ::atoms::type_array.end(),
               cu::atoms::type_array.begin()
               );

         /*
          * Allocate memory and pass the cell information
          */

         cu::atoms::cell_array.resize(::atoms::num_atoms);

         thrust::copy(
               ::atoms::cell_array.begin(),
               ::atoms::cell_array.end(),
               cu::atoms::cell_array.begin()
               );

         /*
          * Allocate the memory for the unrolled spin norm array
          */

         cu::atoms::spin_norm_array.resize(::atoms::num_atoms);

         thrust::copy(
               ::atoms::m_spin_array.begin(),
               ::atoms::m_spin_array.end(),
               cu::atoms::spin_norm_array.begin()
               );

         return true;
      }

      bool __initialize_fields ()
      {
         /*
          * Allocate memory in the device and transfer the
          * total spin field in each atom.
          */

         cu::x_total_spin_field_array.resize(::atoms::num_atoms);
         cu::y_total_spin_field_array.resize(::atoms::num_atoms);
         cu::z_total_spin_field_array.resize(::atoms::num_atoms);

         thrust::copy(
               ::atoms::x_total_spin_field_array.begin(),
               ::atoms::x_total_spin_field_array.end(),
               cu::x_total_spin_field_array.begin()
               );

         thrust::copy(
               ::atoms::y_total_spin_field_array.begin(),
               ::atoms::y_total_spin_field_array.end(),
               cu::y_total_spin_field_array.begin()
               );

         thrust::copy(
               ::atoms::z_total_spin_field_array.begin(),
               ::atoms::z_total_spin_field_array.end(),
               cu::z_total_spin_field_array.begin()
               );

         /*
          * Allocate memory in the device and transfer the
          * total external field in each atom.
          */

         cu::x_total_external_field_array.resize(::atoms::num_atoms);
         cu::y_total_external_field_array.resize(::atoms::num_atoms);
         cu::z_total_external_field_array.resize(::atoms::num_atoms);

         thrust::copy(
               ::atoms::x_total_external_field_array.begin(),
               ::atoms::x_total_external_field_array.end(),
               cu::x_total_external_field_array.begin()
               );

         thrust::copy(
               ::atoms::y_total_external_field_array.begin(),
               ::atoms::y_total_external_field_array.end(),
               cu::y_total_external_field_array.begin()
               );

         thrust::copy(
               ::atoms::z_total_external_field_array.begin(),
               ::atoms::z_total_external_field_array.end(),
               cu::z_total_external_field_array.begin()
               );

         /*
          * Allocate memory and transfer any existing
          * initial data for the dipolar field
          */

         cu::x_dipolar_field_array.resize(::atoms::num_atoms);
         cu::y_dipolar_field_array.resize(::atoms::num_atoms);
         cu::z_dipolar_field_array.resize(::atoms::num_atoms);

         thrust::copy(
               ::atoms::x_dipolar_field_array.begin(),
               ::atoms::x_dipolar_field_array.end(),
               cu::x_dipolar_field_array.begin()
               );

         thrust::copy(
               ::atoms::y_dipolar_field_array.begin(),
               ::atoms::y_dipolar_field_array.end(),
               cu::y_dipolar_field_array.begin()
               );

         thrust::copy(
               ::atoms::z_dipolar_field_array.begin(),
               ::atoms::z_dipolar_field_array.end(),
               cu::z_dipolar_field_array.begin()
               );

         return true;
      }

      bool __initialize_cells ()
      {
         /*
          * Allocate memory and initialize coordinates
          */

         cu::cells::x_coord_array.resize(::cells::num_cells);
         cu::cells::y_coord_array.resize(::cells::num_cells);
         cu::cells::z_coord_array.resize(::cells::num_cells);

         thrust::copy(
               ::cells::x_coord_array.begin(),
               ::cells::x_coord_array.end(),
               cu::cells::x_coord_array.begin()
               );

         thrust::copy(
               ::cells::y_coord_array.begin(),
               ::cells::y_coord_array.end(),
               cu::cells::y_coord_array.begin()
               );

         thrust::copy(
               ::cells::z_coord_array.begin(),
               ::cells::z_coord_array.end(),
               cu::cells::z_coord_array.begin()
               );

         /*
          * Allocate memory and initialize cell magnetization
          */

         cu::cells::x_mag_array.resize(::cells::num_cells);
         cu::cells::y_mag_array.resize(::cells::num_cells);
         cu::cells::z_mag_array.resize(::cells::num_cells);

         thrust::copy(
               ::cells::x_mag_array.begin(),
               ::cells::x_mag_array.end(),
               cu::cells::x_mag_array.begin()
               );

         thrust::copy(
               ::cells::y_mag_array.begin(),
               ::cells::y_mag_array.end(),
               cu::cells::y_mag_array.begin()
               );

         thrust::copy(
               ::cells::z_mag_array.begin(),
               ::cells::z_mag_array.end(),
               cu::cells::z_mag_array.begin()
               );

         /*
          * Allocate memory and initialize cell fields
          */

         cu::cells::x_field_array.resize(::cells::num_cells);
         cu::cells::y_field_array.resize(::cells::num_cells);
         cu::cells::z_field_array.resize(::cells::num_cells);

         thrust::copy(
               ::cells::x_field_array.begin(),
               ::cells::x_field_array.end(),
               cu::cells::x_field_array.begin()
               );

         thrust::copy(
               ::cells::y_field_array.begin(),
               ::cells::y_field_array.end(),
               cu::cells::y_field_array.begin()
               );

         thrust::copy(
               ::cells::z_field_array.begin(),
               ::cells::z_field_array.end(),
               cu::cells::z_field_array.begin()
               );

         /*
          * Copy volume and number of atoms for each cell
          */

         cu::cells::volume_array.resize(::cells::num_cells);

         thrust::copy(
               ::cells::volume_array.begin(),
               ::cells::volume_array.end(),
               cu::cells::volume_array.begin()
               );

         cu::cells::num_atoms.resize(::cells::num_cells);

         thrust::copy(
               ::cells::num_atoms_in_cell.begin(),
               ::cells::num_atoms_in_cell.end(),
               cu::cells::num_atoms.begin()
               );

         return true;
      }

      bool __initialize_materials ()
      {

         /*
          * Serialize material data
          */
         size_t num_mats = ::mp::num_materials;
         thrust::host_vector<material_parameters_t> _materials(num_mats);
         for (size_t i = 0; i < num_mats; i++)
         {
            double mu_s_SI = ::mp::material[i].mu_s_SI;

            _materials[i].alpha =
               ::mp::material[i].alpha;
            _materials[i].gamma_rel =
               ::mp::material[i].gamma_rel;
            _materials[i].mu_s_si =
               mu_s_SI;
            _materials[i].i_mu_s_si =
               1.0 / mu_s_SI;
            _materials[i].k_latt =
               ::mp::material[i].Klatt_SI / mu_s_SI;
            _materials[i].sh2 =
               ::mp::material[i].sh2 / mu_s_SI;
            _materials[i].sh4 =
               ::mp::material[i].sh4 / mu_s_SI;
            _materials[i].sh6 =
               ::mp::material[i].sh6 / mu_s_SI;
            _materials[i].ku =
               ::mp::material[i].Ku;
            _materials[i].anisotropy_unit_x =
               ::mp::material[i].UniaxialAnisotropyUnitVector[0];
            _materials[i].anisotropy_unit_y =
               ::mp::material[i].UniaxialAnisotropyUnitVector[1];
            _materials[i].anisotropy_unit_z =
               ::mp::material[i].UniaxialAnisotropyUnitVector[2];
            _materials[i].applied_field_strength =
               ::mp::material[i].applied_field_strength;
            _materials[i].applied_field_unit_x =
               ::mp::material[i].applied_field_unit_vector[0];
            _materials[i].applied_field_unit_y =
               ::mp::material[i].applied_field_unit_vector[1];
            _materials[i].applied_field_unit_z =
               ::mp::material[i].applied_field_unit_vector[2];
            _materials[i].Kc1_SI =
               ::mp::material[i].Kc1_SI;
            _materials[i].temperature =
               ::mp::material[i].temperature;
            _materials[i].temperature_rescaling_alpha =
               ::mp::material[i].temperature_rescaling_alpha;
            _materials[i].temperature_rescaling_Tc =
               ::mp::material[i].temperature_rescaling_Tc;
            _materials[i].H_th_sigma =
               ::mp::material[i].H_th_sigma;
         }

         /*
          * Allocate memory and send information about the materials
          */
         cu::mp::materials.resize(num_mats);
         thrust::copy(
            _materials.begin(),
            _materials.end(),
            cu::mp::materials.begin()
            );

         return true;
      }

      bool __initialize_topology ()
      {
         /*
          * Send the information for limits and neighbors up to the
          * device.
          *

         // Resize and set all values to 0
         cu::atoms::limits.assign(::atoms::num_atoms + 1UL, 0);
         cu::atoms::neighbours.resize(::atoms::total_num_neighbours);

         thrust::copy(
               ::atoms::neighbour_list_end_index.begin(),
               ::atoms::neighbour_list_end_index.end(),
               cu::atoms::limits.begin() + 1UL
               );

         *
          * Transform the limits to be one pased the last element
          * in the neighbors list.
          *
         thrust::transform(
               cu::atoms::limits.begin(),
               cu::atoms::limits.end(),
               cu::atoms::limits.begin(),
               cu::plusone_functor()
               );

         thrust::copy(
               ::atoms::neighbour_list_array.begin(),
               ::atoms::neighbour_list_array.end(),
               cu::atoms::neighbours.begin()
               );
         */

         // Transfer the row ptrs and col indices to the device
         std::vector<int> limits_h( ::atoms::num_atoms + 1, 0);
         for( int atom = 0; atom < ::atoms::num_atoms; atom++)
            limits_h[atom+1] = ::atoms::neighbour_list_end_index[atom]+1;

         cu::atoms::limits.resize( ::atoms::num_atoms + 1);
         cu::atoms::neighbours.resize( ::atoms::neighbour_list_array.size() );


         thrust::copy(
               limits_h.begin(),
               limits_h.end(),
               cu::atoms::limits.begin()
               );

         thrust::copy(
               ::atoms::neighbour_list_array.begin(),
               ::atoms::neighbour_list_array.end(),
               cu::atoms::neighbours.begin()
               );



         return true;
      }

      bool __initialize_curand ()
      {
         hipMalloc (
               (void **) &cu::d_rand_state,
               cu::grid_size * cu::block_size * sizeof(hiprandState));

         check_cuda_errors (__FILE__, __LINE__);

         cu::init_rng <<< cu::grid_size, cu::block_size >>> (
               cu::d_rand_state, ::mtrandom::integration_seed);

         check_cuda_errors (__FILE__, __LINE__);

         return true;
      }

      bool __initialize_stats ()
      {
         std::vector<int> mask;
         std::vector<double> saturations;

         ::stats::system_magnetization.get_mask(mask, saturations);
         cu::stats::system_mask_size = saturations.size();
         cu::stats::system_mask.resize(mask.size());
         thrust::copy (
               mask.begin(),
               mask.end(),
               cu::stats::system_mask.begin()
               );
         cu::stats::system_magnetization.resize(4 * saturations.size());
         cu::stats::system_mean_magnetization.resize(4 * saturations.size());
         check_cuda_errors (__FILE__, __LINE__);

         ::stats::material_magnetization.get_mask(mask, saturations);
         cu::stats::material_mask_size = saturations.size();
         cu::stats::material_mask.resize(mask.size());
         thrust::copy (
               mask.begin(),
               mask.end(),
               cu::stats::material_mask.begin()
               );
         cu::stats::material_magnetization.resize(4 * saturations.size());
         cu::stats::material_mean_magnetization.resize(4 * saturations.size());
         check_cuda_errors (__FILE__, __LINE__);

         ::stats::height_magnetization.get_mask(mask, saturations);
         cu::stats::height_mask_size = saturations.size();
         cu::stats::height_mask.resize(mask.size());
         thrust::copy (
               mask.begin(),
               mask.end(),
               cu::stats::height_mask.begin()
               );
         cu::stats::height_magnetization.resize(4 * saturations.size());
         cu::stats::height_mean_magnetization.resize(4 * saturations.size());
         check_cuda_errors (__FILE__, __LINE__);

         ::stats::material_height_magnetization.get_mask(mask, saturations);
         cu::stats::material_height_mask_size = saturations.size();
         cu::stats::material_height_mask.resize(mask.size());
         thrust::copy (
               mask.begin(),
               mask.end(),
               cu::stats::material_height_mask.begin()
               );
         cu::stats::material_height_magnetization.resize(4 * saturations.size());
         cu::stats::material_height_mean_magnetization.resize(4 * saturations.size());
         check_cuda_errors (__FILE__, __LINE__);

         return true;

      }

      /**
       * Inits the random number generator states in the device, one per thread
       */
      __global__ void init_rng (hiprandState * states, int seed)
      {
         int tid = blockIdx.x * blockDim.x + threadIdx.x;
         hiprand_init (seed, tid, 0, &states[tid]);
      }
   }

#endif

} // end of namespace vcuda
