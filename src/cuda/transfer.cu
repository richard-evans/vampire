//------------------------------------------------------------------------------
//
//   This file is part of the VAMPIRE open source package under the
//   Free BSD licence (see licence file for details).
//
//   (c) Richard F L Evans 2019. All rights reserved.
//
//------------------------------------------------------------------------------
//
// C++ standard library headers

// Vampire headers
#include "cuda.hpp"


// Local cuda headers
#include "cuda_utils.hpp"
#include "data.hpp"
#include "dipole.hpp"
#include "internal.hpp"
#include "typedefs.hpp"

// Conditional compilation of all cuda code
#ifdef CUDA

// namespace aliasing for brevity
namespace cu = vcuda::internal;

// vampire cuda namespace
namespace vcuda{

//------------------------------------------------------------------------------
// Wrapper function to transfer spin data from GPU to CPU
//------------------------------------------------------------------------------
void transfer_spin_positions_from_gpu_to_cpu(){

   hipMemcpy(internal::h_x_spin_transfer_buffer, internal::atoms::d_x_spin, ::atoms::num_atoms * sizeof(cu::cu_real_t), hipMemcpyDeviceToHost);
   hipMemcpy(internal::h_y_spin_transfer_buffer, internal::atoms::d_y_spin, ::atoms::num_atoms * sizeof(cu::cu_real_t), hipMemcpyDeviceToHost);
   hipMemcpy(internal::h_z_spin_transfer_buffer, internal::atoms::d_z_spin, ::atoms::num_atoms * sizeof(cu::cu_real_t), hipMemcpyDeviceToHost);

   std::copy(internal::h_x_spin_transfer_buffer, internal::h_x_spin_transfer_buffer + ::atoms::num_atoms, ::atoms::x_spin_array.begin());
   std::copy(internal::h_y_spin_transfer_buffer, internal::h_y_spin_transfer_buffer + ::atoms::num_atoms, ::atoms::y_spin_array.begin());
   std::copy(internal::h_z_spin_transfer_buffer, internal::h_z_spin_transfer_buffer + ::atoms::num_atoms, ::atoms::z_spin_array.begin());
   /*
   thrust::copy(internal::atoms::x_spin_array.begin(),internal::atoms::x_spin_array.end(),::atoms::x_spin_array.begin());
   thrust::copy(internal::atoms::y_spin_array.begin(),internal::atoms::y_spin_array.end(),::atoms::y_spin_array.begin());
   thrust::copy(internal::atoms::z_spin_array.begin(),internal::atoms::z_spin_array.end(),::atoms::z_spin_array.begin());
   */
   return;

}

//------------------------------------------------------------------------------
// Wrapper function to transfer atomic dipole field data from CPU to GPU
//------------------------------------------------------------------------------
void transfer_dipole_fields_from_cpu_to_gpu(){

   size_t num_bytes = ::atoms::num_atoms * sizeof(cu::cu_real_t);
   std::vector<cu::cu_real_t> tmp_buffer;
   tmp_buffer.resize(::atoms::num_atoms);

   std::copy(::dipole::atom_dipolar_field_array_x.begin(), ::dipole::atom_dipolar_field_array_x.end(), tmp_buffer.begin());
   hipMemcpy(cu::d_x_dip_field, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);
   std::copy(::dipole::atom_dipolar_field_array_y.begin(), ::dipole::atom_dipolar_field_array_y.end(), tmp_buffer.begin());
   hipMemcpy(cu::d_y_dip_field, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);
   std::copy(::dipole::atom_dipolar_field_array_z.begin(), ::dipole::atom_dipolar_field_array_z.end(), tmp_buffer.begin());
   hipMemcpy(cu::d_z_dip_field, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);

   std::copy(::dipole::atom_mu0demag_field_array_x.begin(), ::dipole::atom_mu0demag_field_array_x.end(), tmp_buffer.begin());
   hipMemcpy(cu::d_x_mu0H_dip_field, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);
   std::copy(::dipole::atom_mu0demag_field_array_y.begin(), ::dipole::atom_mu0demag_field_array_y.end(), tmp_buffer.begin());
   hipMemcpy(cu::d_y_mu0H_dip_field, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);
   std::copy(::dipole::atom_mu0demag_field_array_z.begin(), ::dipole::atom_mu0demag_field_array_z.end(), tmp_buffer.begin());
   hipMemcpy(cu::d_z_mu0H_dip_field, tmp_buffer.data(), num_bytes, hipMemcpyHostToDevice);
   
   /*
   thrust::copy(::dipole::atom_dipolar_field_array_x.begin(),::dipole::atom_dipolar_field_array_x.end(), cu::x_dipolar_field_array.begin());
   thrust::copy(::dipole::atom_dipolar_field_array_y.begin(),::dipole::atom_dipolar_field_array_y.end(), cu::y_dipolar_field_array.begin());
   thrust::copy(::dipole::atom_dipolar_field_array_z.begin(),::dipole::atom_dipolar_field_array_z.end(), cu::z_dipolar_field_array.begin());
   */
   return;

}

//------------------------------------------------------------------------------
// Wrapper function to transfer dipole field data from GPU to CPU
//------------------------------------------------------------------------------
void transfer_dipole_cells_fields_from_gpu_to_cpu(){

   hipMemcpy(::dipole::cells_field_array_x.data(), cu::cells::d_x_cell_field, ::cells::num_cells * sizeof(double), hipMemcpyDeviceToHost);
   hipMemcpy(::dipole::cells_field_array_y.data(), cu::cells::d_y_cell_field, ::cells::num_cells * sizeof(double), hipMemcpyDeviceToHost);
   hipMemcpy(::dipole::cells_field_array_z.data(), cu::cells::d_z_cell_field, ::cells::num_cells * sizeof(double), hipMemcpyDeviceToHost);

   hipMemcpy(::dipole::cells_mu0Hd_field_array_x.data(), cu::cells::d_x_cell_mu0H_field, ::cells::num_cells * sizeof(double), hipMemcpyDeviceToHost);
   hipMemcpy(::dipole::cells_mu0Hd_field_array_y.data(), cu::cells::d_y_cell_mu0H_field, ::cells::num_cells * sizeof(double), hipMemcpyDeviceToHost);
   hipMemcpy(::dipole::cells_mu0Hd_field_array_z.data(), cu::cells::d_z_cell_mu0H_field, ::cells::num_cells * sizeof(double), hipMemcpyDeviceToHost);
    
   return;

}

} // end of vcuda namespace

#endif
