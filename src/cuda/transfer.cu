//------------------------------------------------------------------------------
//
//   This file is part of the VAMPIRE open source package under the
//   Free BSD licence (see licence file for details).
//
//   (c) Richard F L Evans 2019. All rights reserved.
//
//------------------------------------------------------------------------------
//
// C++ standard library headers

// Vampire headers
#include "cuda.hpp"


// Local cuda headers
#include "cuda_utils.hpp"
#include "data.hpp"
#include "dipole.hpp"
#include "internal.hpp"
#include "typedefs.hpp"

// Conditional compilation of all cuda code
#ifdef CUDA

// namespace aliasing for brevity
namespace cu = vcuda::internal;

// vampire cuda namespace
namespace vcuda{

//------------------------------------------------------------------------------
// Wrapper function to transfer spin data from GPU to CPU
//------------------------------------------------------------------------------
void transfer_spin_positions_from_gpu_to_cpu(){

   hipMemcpy(::atoms::x_spin_array.data(), cu::atoms::d_x_spin, ::atoms::num_atoms * sizeof(cu::cu_real_t), hipMemcpyDeviceToHost);
   hipMemcpy(::atoms::y_spin_array.data(), cu::atoms::d_y_spin, ::atoms::num_atoms * sizeof(cu::cu_real_t), hipMemcpyDeviceToHost);
   hipMemcpy(::atoms::z_spin_array.data(), cu::atoms::d_z_spin, ::atoms::num_atoms * sizeof(cu::cu_real_t), hipMemcpyDeviceToHost);
   
   /*
   thrust::copy(internal::atoms::x_spin_array.begin(),internal::atoms::x_spin_array.end(),::atoms::x_spin_array.begin());
   thrust::copy(internal::atoms::y_spin_array.begin(),internal::atoms::y_spin_array.end(),::atoms::y_spin_array.begin());
   thrust::copy(internal::atoms::z_spin_array.begin(),internal::atoms::z_spin_array.end(),::atoms::z_spin_array.begin());
   */
   return;

}

//------------------------------------------------------------------------------
// Wrapper function to transfer dipole field data from CPU to GPU
//------------------------------------------------------------------------------
void transfer_dipole_fields_from_cpu_to_gpu(){

   hipMemcpy(::dipole::atom_dipolar_field_array_x.data(), cu::d_x_dip_field, ::atoms::num_atoms * sizeof(cu::cu_real_t), hipMemcpyHostToDevice);
   hipMemcpy(::dipole::atom_dipolar_field_array_y.data(), cu::d_y_dip_field, ::atoms::num_atoms * sizeof(cu::cu_real_t), hipMemcpyHostToDevice);
   hipMemcpy(::dipole::atom_dipolar_field_array_z.data(), cu::d_z_dip_field, ::atoms::num_atoms * sizeof(cu::cu_real_t), hipMemcpyHostToDevice);

   hipMemcpy(::dipole::atom_mu0demag_field_array_x.data(), cu::d_x_mu0H_dip_field, ::atoms::num_atoms * sizeof(cu::cu_real_t), hipMemcpyHostToDevice);
   hipMemcpy(::dipole::atom_mu0demag_field_array_y.data(), cu::d_y_mu0H_dip_field, ::atoms::num_atoms * sizeof(cu::cu_real_t), hipMemcpyHostToDevice);
   hipMemcpy(::dipole::atom_mu0demag_field_array_z.data(), cu::d_z_mu0H_dip_field, ::atoms::num_atoms * sizeof(cu::cu_real_t), hipMemcpyHostToDevice);

   /*
   thrust::copy(::dipole::atom_dipolar_field_array_x.begin(),::dipole::atom_dipolar_field_array_x.end(), cu::x_dipolar_field_array.begin());
   thrust::copy(::dipole::atom_dipolar_field_array_y.begin(),::dipole::atom_dipolar_field_array_y.end(), cu::y_dipolar_field_array.begin());
   thrust::copy(::dipole::atom_dipolar_field_array_z.begin(),::dipole::atom_dipolar_field_array_z.end(), cu::z_dipolar_field_array.begin());
   */
   return;

}

} // end of vcuda namespace

#endif
