//------------------------------------------------------------------------------
//
//   This file is part of the VAMPIRE open source package under the
//   Free BSD licence (see licence file for details).
//
//   (c) Richard F L Evans 2019. All rights reserved.
//
//------------------------------------------------------------------------------
//
// C++ standard library headers

// Vampire headers
#include "cuda.hpp"

// Local cuda headers
#include "cuda_utils.hpp"
#include "data.hpp"
#include "dipole.hpp"
#include "internal.hpp"

// Conditional compilation of all cuda code
#ifdef CUDA

// namespace aliasing for brevity
namespace cu = vcuda::internal;

// vampire cuda namespace
namespace vcuda{

//------------------------------------------------------------------------------
// Wrapper function to transfer spin data from GPU to CPU
//------------------------------------------------------------------------------
void transfer_spin_positions_from_gpu_to_cpu(){

   thrust::copy(internal::atoms::x_spin_array.begin(),internal::atoms::x_spin_array.end(),::atoms::x_spin_array.begin());
   thrust::copy(internal::atoms::y_spin_array.begin(),internal::atoms::y_spin_array.end(),::atoms::y_spin_array.begin());
   thrust::copy(internal::atoms::z_spin_array.begin(),internal::atoms::z_spin_array.end(),::atoms::z_spin_array.begin());

   return;

}

//------------------------------------------------------------------------------
// Wrapper function to transfer dipole field data from CPU to GPU
//------------------------------------------------------------------------------
void transfer_dipole_fields_from_cpu_to_gpu(){

   thrust::copy(::dipole::atom_dipolar_field_array_x.begin(),::dipole::atom_dipolar_field_array_x.end(), cu::x_dipolar_field_array.begin());
   thrust::copy(::dipole::atom_dipolar_field_array_y.begin(),::dipole::atom_dipolar_field_array_y.end(), cu::y_dipolar_field_array.begin());
   thrust::copy(::dipole::atom_dipolar_field_array_z.begin(),::dipole::atom_dipolar_field_array_z.end(), cu::z_dipolar_field_array.begin());

   return;

}

} // end of vcuda namespace

#endif
